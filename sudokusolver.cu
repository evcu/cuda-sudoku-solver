#include "hip/hip_runtime.h"
/*
 *  Please write your name and net ID below
 *
 *  Last name: Evci
 *  First name: Utku
 *  Net ID: ue225
 *
 */


/*
 * You compile with:
 * 		nvcc -arch sm_35 -rdc=true -o sudokusolver sudokusolver.cu
 */
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
// #define DEBUG

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
/*****************************************************************/

__global__ void fillSudokuSafeAndFork(char* memory,int* stats)
{
  /*
    We have nBlocks many available blocks.
  */
  uint i,current_poss,j,temp,mat_i,mat_j,k;
  char* block_memory = memory+(81*blockIdx.x);

  __shared__ uint row_used_numbers[9];
  __shared__ uint col_used_numbers[9];
  __shared__ uint cell_used_numbers[9];
  __shared__ char progress_flag;
  __shared__ char done_flag;
  __shared__ char error_flag;
  __shared__ int min_forks;
  __shared__ int scheduling_thread;

    // check whether all blocks are idle or not.This shouldn't happen
    if (blockIdx.x==0){ //first block
      if (threadIdx.x==0) {
        progress_flag=0;
      }
      __syncthreads();
      for(i=threadIdx.x;i<gridDim.x;i+=blockDim.x){
        if (stats[i] > 0){
          progress_flag=1;
        }
      }
      __syncthreads();
      if (progress_flag==0 and threadIdx.x == 0){
        //no active block terminate.
        if (threadIdx.x==0) printf("no active blocks...terminating\n");
        stats[gridDim.x]=2;
      }
    }

    //If block is active work on it.
    if(stats[blockIdx.x]==1){
      if (threadIdx.x==0) {
        error_flag = 0;  //set to 1 if the board is wrongs
        done_flag = 0;  //set to 1 if the board is solved
        progress_flag=1; //set to 0 if no reterministic progress can be made.
      }
      __syncthreads();

      while(!error_flag && !done_flag &&progress_flag ){
        __syncthreads();
        // 1st check whether the board is valid and fill X_used_numbers arrays for rows,columns and cells.
        //*************************
        if (threadIdx.x<9){
          // TODO optimize here such that there wraps does row/col/cell.
            row_used_numbers[threadIdx.x] = 0;
            col_used_numbers[threadIdx.x] = 0;
            cell_used_numbers[threadIdx.x] = 0;
            for(i=0;i<9;i++){
                //rows
                temp = block_memory[threadIdx.x*9+i];
                if (temp) { //!=0
                    if ((row_used_numbers[threadIdx.x]>>(temp-1)) & 1){
                        // This is bad, you have the same number in the same row. This solution fails
                        error_flag=10+i;
                    }
                    //set n'th bit to 1.
                    row_used_numbers[threadIdx.x] |= 1<<(temp-1);
                }
                //columns
                temp = block_memory[i*9+threadIdx.x];
                if (temp) { //!=0
                    if ((col_used_numbers[threadIdx.x]>>(temp-1)) & 1){
                        // This is bad, you have the same number in the same column. This solution fails
                        error_flag=20+i;
                    }
                    //set n'th bit to 1.
                    col_used_numbers[threadIdx.x] |= 1<<(temp-1);
                }
            }
            //cells
            for (i=(threadIdx.x/3)*3;i<((threadIdx.x/3+1)*3);i++){
              for (j=(threadIdx.x%3)*3;j<((threadIdx.x%3+1)*3);j++){
                temp = block_memory[i*9+j];
                if (temp) { //!=0
                    if ((cell_used_numbers[threadIdx.x]>>(temp-1)) & 1){
                        // This is bad, you have the same number in the same cell. This solution fails
                        error_flag=30+i;
                    }
                    //set n'th bit to 1.
                    cell_used_numbers[threadIdx.x] |= 1<<(temp-1);
                }
              }
            }

            }
        __syncthreads();
        if (error_flag==0){
            if (threadIdx.x==0) {
              progress_flag = 0;
              done_flag = 1;
            }
            __syncthreads();
           if (threadIdx.x<81){
                // 2nd for each cell calculate available numbers(row_used OR col_used OR cell_used) and if there is one 0
                //*************************
                current_poss = 0;
                mat_i = threadIdx.x/9;
                mat_j = threadIdx.x%9;
                if (block_memory[threadIdx.x] == 0){
                    done_flag = 0;
                    current_poss = (row_used_numbers[mat_i] | col_used_numbers[mat_j] | cell_used_numbers[(mat_i/3)*3+(mat_j/3)]);
                    //printf("thredix=%d,current_poss=%d\n",threadIdx.x,current_poss);
                    temp = 0; // temp for count
                    for (i=0;i<9;i++){
                      if ((current_poss & (1<<i))==0){
                        if (temp){ //if there is a zero found already
                          temp = 10;
                          break;
                        }
                        else{
                          temp = i+1;
                        }
                      }
                    }
                    if (temp==0){
                        #ifdef DEBUG
                          printf("Block:%d,i=%d,j=%d, cannot be filled. Invalidating\n",blockIdx.x,mat_i+1,mat_j+1);
                        #endif
                        error_flag = 1;
                        progress_flag = 1;
                    }
                    else if (temp<=9){
                      #ifdef DEBUG
                      if (blockIdx.x==0)  printf("i=%d,j=%d,val=%d\n",threadIdx.x/9+1,(threadIdx.x%9)+1,temp);
                      #endif
                      block_memory[threadIdx.x] = temp;
                      progress_flag = 1;
                    }
                }
            }
          }
          __syncthreads();
        }
        /*End of while loop, lets see why the while loop is ended.
          error: we make stats[.]=0 to make the block available for future scheduling.
          done: set the extra stat value to 2.
          no_progress: pick the less ambigous cell and fork. For each extra possibility one of the threads find an available block using atomic operations.
          */
        __syncthreads();
        if (done_flag) {
          if (threadIdx.x==0){
            memcpy(memory+gridDim.x*81,block_memory,81);
            stats[gridDim.x]=2;
          }
        }
        else if (error_flag!=0){
          #ifdef DEBUG
            if (threadIdx.x==0)
              printf("There is an error:%d : with this block %d \n",error_flag,blockIdx.x);
          #endif
          if (threadIdx.x==0)
            stats[blockIdx.x]=0;
        }

        else if (progress_flag==0) {
          // Implement scheduling here
          if (threadIdx.x==0){
            min_forks = 9;
            scheduling_thread = blockDim.x; //bigger then and threadID
          }
          __syncthreads();
          //lets find out what is the minimum number of possible digits for any cell.
          temp = 0; // temp for count
          if(current_poss!=0){
            for (i=0;i<9;i++){
              if ((current_poss & (1<<i))==0){
                  temp++;
                }
            }
            atomicMin(&min_forks,temp);
          }
          __syncthreads();
          //then out of most ambigous cells lets choose the one with smallest threadIdx.
          if (temp==min_forks){
            atomicMin(&scheduling_thread,threadIdx.x);
          }
          __syncthreads();
          if (scheduling_thread==threadIdx.x){
            //Find a suitable block to schedule the fork for each extra value.
            k = 1;
            j=0; //to continue wherever we stayed on the previous search.
            for (i=0;i<9;i++){
              if ((current_poss & (1<<i))==0){
                    if (k==1) {
                      // first possibility stays with the current block
                      block_memory[threadIdx.x] = i+1;
                    }
                    else{
                      // look for suitable block
                      for (;j<gridDim.x;j++) {
                        atomicCAS(stats+j,0,gridDim.x*blockIdx.x+threadIdx.x+2); //unique identifier>1
                        if (stats[j] == (gridDim.x*blockIdx.x+threadIdx.x+2)){
                          //succesful scheduling
                          #ifdef DEBUG
                          if (blockIdx.x<5){
                            printf("Error_flag:%d,current_poss=%d,rowi=%d\n",error_flag,current_poss,row_used_numbers[mat_i]);
                             printf("From-Block:%d,Forkedinto: Block:%d,i=%d,j=%d,new_val=%d,k=%d/%d\n",blockIdx.x,j,mat_i+1,mat_j+1,i+1,k,min_forks);
                          }
                           #endif
                          memcpy(memory+j*81,block_memory,81);
                          memory[j*81+threadIdx.x] = i+1;
                          stats[j] = 1;
                          break;
                        }
                      }
                      if (j == gridDim.x){
                         // #ifdef DEBUG
                        printf("From-Block:%d,i=%d,j=%d,k=%d/%d,c_poss=%d,b_mem=%d,chouldnt schedule\n",blockIdx.x,mat_i+1,mat_j+1,k,min_forks,current_poss,block_memory[threadIdx.x]);
                         // #endif
                      }
                    }
                    k++;
                }
            }
          }
          __syncthreads();
        }
    }
  }


__device__ void printSudokuDev(char *arr){
  int i;
  printf("|++++++++++++++++++++");
  for (i = 0;i<81;i++){
    if (i%27==0) printf("|\n|--------------------");
    if (i%9==0) printf("|\n");
    if (i%3==0) printf("|");
    if (arr[i]){
      printf("%d ",arr[i]);
    }
    else{
      printf(". ");
    }
  }
  printf("|\n");
}

__global__ void controller(char* arr_dev,int* block_stat,int nBlocks, int nThreads){
  int  i = 0;
  while (block_stat[nBlocks]!=2 && i<55){//stats[gridDim.x]==2 means, solution is coppied to the last 81 char of memory.
    fillSudokuSafeAndFork<<<nBlocks,nThreads>>>(arr_dev,block_stat);
    hipDeviceSynchronize();
    #ifdef DEBUG
      printf("Hey\n");
      for (int j=0;j=nBlocks;j++){
        printf("%d,",block_stat[j]);
      }
      printf("\n");
      printSudokuDev(arr_dev);
    #endif
    i++;
  }
}

void  gpu_sudoku_solver(char* arr,int version)
{
  char *arr_dev;
  int *block_stat;

  int nThreads = 96; // wrap_size 32, each thread will have responsible from one cell.
  int nBlocks = 20000; // max available concurent blocks/searches running.
  int memSize = 81*(nBlocks+1); // 0.81 MB for N=9
  //copy array and create a new one temp. last block/stat is for the result
  hipMalloc((void**) &block_stat,(nBlocks+1)*sizeof(int));
  hipMemset(block_stat, 0, (nBlocks+1)*sizeof(int));
  hipMemset(block_stat, 1, 1);
  if( !block_stat )
  {
   fprintf(stderr, " Cannot allocate block_stat array of size %d on the device\n", (nBlocks+1)*sizeof(int));
   exit(1);
  }

  hipMalloc((void**) &arr_dev,memSize);
  hipMemcpy(arr_dev,arr,81,hipMemcpyHostToDevice);

  if( !arr_dev )
  {
   fprintf(stderr, " Cannot allocate arr_dev of size %d on the device\n", memSize);
   exit(1);
  }

  if (version==1){
    printf("Block=%d,threads=%d starting\n",nBlocks,nThreads);
    controller<<<1,1>>>(arr_dev,block_stat,nBlocks,nThreads);
    hipError_t err = hipGetLastError();
      if (err != hipSuccess)
          printf("Error: %s\n", hipGetErrorString(err));
  }
  else{
    printf("Invalid version'\n");
    exit(1);
  }
  hipMemcpy(arr,arr_dev+81*nBlocks,81,hipMemcpyDeviceToHost);
  hipFree(arr_dev);
  hipFree(block_stat);
}

void readNextSudoku(FILE *fp, char *sudoku){
  int i,j;
  for (i=0;i<9;i++){
    fscanf(fp, "%s", sudoku);
    for (j=0;j<9;j++){
      sudoku[j] = sudoku[j]-48; //48 = char '0'
    }
    sudoku = sudoku + 9;
  }
}

void printSudoku(char *arr){
  int i;
  printf("|++++++++++++++++++++");
  for (i = 0;i<81;i++){
    if (i%27==0) printf("|\n|--------------------");
    if (i%9==0) printf("|\n");
    if (i%3==0) printf("|");
    if (arr[i]){
      printf("%d ",arr[i]);
    }
    else{
      printf(". ");
    }
  }
  printf("|\n");
}

void printSudokuToFile(char *arr,FILE* out_fp){
  for (int i = 0;i<81;i++){
    fprintf(out_fp, "%d",arr[i]);
    if (i%9==8){
      fprintf(out_fp, "\n");
    }
  }
  fprintf(out_fp, "\n");
}

int main(int argc, char *argv[])
{
  hipFree(0);
  hipSetDevice ( 1 );
  int version=1,flag=1,i;
  char cur_sudoku[81];
  char out_name[200];
  double time_taken;
  clock_t start, end;
  FILE *inp_fp,*out_fp;

  if (argc <=3 && argc>1  )
    {
      memset(out_name, '\0', sizeof(out_name));
      strcpy(out_name, argv[1]);
      i = 0 ;
      while (out_name[i] != '\0' && out_name[i] != '.'){
        i ++;
      }
      if (out_name[i] == '\0'){
        fprintf(stderr, "The input file should be like X.in: %s!\n",out_name[i]);
        exit(1);
      }
      else{
        out_name[i+1] = 's';
        out_name[i+2] = 'o';
        out_name[i+3] = 'l';
        out_name[i+4] = '\0';
      }
      out_fp = fopen(out_name,"w");
      if (out_fp == NULL) {
        fprintf(stderr, "Can't open output file %s!\n",out_name);
        exit(1);
      }

      inp_fp = fopen(argv[1],"r");
      if (inp_fp == NULL) {
        fprintf(stderr, "Can't open input file %s!\n",argv[1]);
        exit(1);
      }
    }
  else
    {
      printf("Usage 'sudokusolver filename [version_no=1]'\n");
      exit(1);
    }

  if (argc == 3 ){
    version = (int) strtol(argv[2], (char **)NULL, 10);
    printf("Version = %d\n", version);
  }
    while (flag!=-1){
      readNextSudoku(inp_fp,cur_sudoku);
      printSudoku(cur_sudoku);
      start = clock();
      gpu_sudoku_solver(cur_sudoku,version);
      end = clock();
      time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
      printf("Time taken for the function_call is %lf\n", time_taken);
      printSudoku(cur_sudoku);
      printf("-----------------------------------------\n");
      printSudokuToFile(cur_sudoku,out_fp);
      fgetc( inp_fp );
      flag=fgetc( inp_fp );
    }
    fclose(inp_fp);
    fclose(out_fp);
}
